/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */

#include "cudaMemoryController.h"
#include <hip/hip_runtime.h>
#include <iostream>

cudaMemoryController::cudaMemoryController()
{
    Data = NULL;
    DataSize = 0;
}
cudaMemoryController::~cudaMemoryController() {
    freeCudaMemory();
}

int* cudaMemoryController::cudaHostAllocMemory(unsigned int DataSize)
{
    std::cout << "Data ? NULL" << DataSize << std::endl;
    if(Data != NULL)
    {
        std::cout << "Data ! NULL" << std::endl;
        freeCudaMemory();
        Data = NULL;
    }
    std::cout << "hipHostAlloc" << std::endl;
    if (hipSuccess != hipHostAlloc((void**)&Data, DataSize*sizeof(int), hipHostMallocWriteCombined | hipHostMallocMapped))
    {
        std::cout << "cudaNotSuccess" << std::endl;
        Data = NULL;
        return Data;
    }
    this->DataSize = DataSize;
    std::cout << "return Data" << std::endl;
    return Data;
}

void cudaMemoryController::freeCudaMemory()
{
    hipFree(Data);
}