#include "hip/hip_runtime.h"
#include "master.h"

void Master::Run(int ThreadID)
{
  hipSetDevice(ThreadID-1);
  hipSetDeviceFlags(hipDeviceMapHost);
  SyncBlock.Barier(ThreadID);
  int* HalfData=ThreadID % 2 ? Data : Data + DataSize / 2;
  int HalfDataSize=ThreadID % 2 ? DataSize / 2 : (DataSize % 2 ? DataSize / 2 + 1 : DataSize / 2);
  for (int i = 1; i <= N; i++)
  {
    <<< x, y >>>Counting(HalfData, HalfDataSize, i);
    hipDeviceSynchronize();
    SyncBlock.Barier(ThreadID);
  }
}

__global__ Counting(int* HalfData, int HalfDataSize, int N)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<HalfDataSize)
  {
    HalfData[i]*=N;
  }
}
