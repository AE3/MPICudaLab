#include "hip/hip_runtime.h"
#include "master.h"

#define THREAD 1536
__host__ void Master::Run(int ThreadID)
{
  hipSetDevice(ThreadID-1);
  hipSetDeviceFlags(hipDeviceMapHost);
  SyncBlock.Barier(ThreadID);
  int* DevData;
  int* HalfData=ThreadID % 2 ? Data : Data + DataSize / 2;
  hipHostGetDevicePointer(&DevData,HalfData,0);
  int HalfDataSize=ThreadID % 2 ? DataSize / 2 : (DataSize % 2 ? DataSize / 2 + 1 : DataSize / 2);
  dim3 block_dim(THREAD);
	dim3 grid_dim((HalfDataSize + block_dim.x - 1) / block_dim.x);
  for (int i = 1; i <= N; i++)
  {
    <<< block_dim, grid_dim >>>Counting(HalfData, HalfDataSize, i);
    hipDeviceSynchronize();
    SyncBlock.Barier(ThreadID);
  }
}

__global__ Counting(int* HalfData, int HalfDataSize, int N)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<HalfDataSize)
  {
    HalfData[i]*=N;
  }
}

__host__ void Master::ReadFile(std::string FileName)
{
  std::ifstream in(FileName);
  in>>DataSize>>N;
  hipHostAlloc((void**)&Data,DataSize*sizeof(int),hipHostMallocMapped);
  for(unsigned int i=0; i<DataSize; i++)
  {
    in>>Data[i];
  }
}
