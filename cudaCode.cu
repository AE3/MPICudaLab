#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */

#include <hip/hip_runtime.h>
#include <cudaCode.h>
#include <iostream>
#include <fstream>

#define THREAD 1536

__global__ void Counting(int* HalfData, int HalfDataSize, int N)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<HalfDataSize)
	{
		HalfData[i] *= N;
	}
}

__host__ void cudaSets(int threadID)
{
    hipSetDevice(threadID);
    hipSetDeviceFlags(hipDeviceMapHost);
}

__host__ void cudaRun(int* Data, unsigned int DataSize,int i)
{
    dim3 blockDim(THREAD);
    dim3 gridDim((DataSize + blockDim.x - 1) / blockDim.x);
    std::cout<<"start count"<<std::endl;
    Counting <<<blockDim, gridDim>>> (Data, DataSize, i);
    hipDeviceSynchronize();
}



__host__ int* setCudaData(int* CData)
{
    int* Data;
    if(hipSuccess != hipHostGetDevicePointer(&Data, CData,0))
    {
        std::cout<<"hipHostGetDevicePointer error"<<std::endl;
    }
    std::cout<< Data << " " << CData[0] <<std::endl;
    return Data;
}