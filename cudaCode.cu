#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */

#include <hip/hip_runtime.h>
#include <cudaCode.h>
#include <iostream>
#include <fstream>

#define THREAD 1536

__global__ void Counting(int* HalfData, int HalfDataSize, int N)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<HalfDataSize)
	{
		HalfData[i] *= N;
	}
}

__host__ void cudaSets(int threadID)
{
    hipSetDevice(threadID);
    hipSetDeviceFlags(hipDeviceMapHost);
}

__host__ void cudaRun(int* Data, unsigned int DataSize,int i)
{
    dim3 blockDim(THREAD);
    dim3 gridDim((DataSize + blockDim.x - 1) / blockDim.x);
    std::cout<<"start count"<<std::endl;
    Counting <<<blockDim, gridDim>>> (Data, DataSize, i);
    out<<std::endl;
    hipDeviceSynchronize();
}



__host__ void setCudaData(int* Data, int* CData)
{
    hipHostGetDevicePointer(&Data, CData,0);
}