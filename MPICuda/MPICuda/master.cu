#include "hip/hip_runtime.h"
#define THREAD 1536

#include "master.h"
#include <fstream>
#include "hip/hip_runtime.h"
#include ""

__global__ void Counting(int* HalfData, int HalfDataSize, int N)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<HalfDataSize)
	{
		HalfData[i] *= N;
	}
}

__host__ void Master::Run(int ThreadID)
{
  hipSetDevice(ThreadID-1);
  hipSetDeviceFlags(hipDeviceMapHost);
  SyncBlock.Barier(ThreadID);
  int* DevData;
  int* HalfData=ThreadID % 2 ? Data : Data + DataSize / 2;
  hipHostGetDevicePointer(&DevData,HalfData,0);
  int HalfDataSize=ThreadID % 2 ? DataSize / 2 : (DataSize % 2 ? DataSize / 2 + 1 : DataSize / 2);
  dim3 block_dim(THREAD);
	dim3 grid_dim((HalfDataSize + block_dim.x - 1) / block_dim.x);
  for (int i = 1; i <= N; i++)
  {
	Counting << <block_dim, grid_dim >> > (HalfData, HalfDataSize, i);
    hipDeviceSynchronize();
    SyncBlock.Barier(ThreadID);
  }
}



__host__ void Master::ReadFile(std::string FileName)
{
  std::ifstream in(FileName);
  in>>DataSize>>N;
  if (hipSuccess != hipHostAlloc((void**)&Data, DataSize*sizeof(int), hipHostMallocWriteCombined | hipHostMallocMapped))
  {
	  return;
  }
  for(unsigned int i=0; i<DataSize; i++)
  {
    in>>Data[i];
  }
}
